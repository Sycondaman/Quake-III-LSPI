#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "TestBlas.h"
#include "blas.h"
#include "Matrix.h"
#include <cmath>
#include <hipblas.h>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>

#define Threshold 0.01f

bool TestBlas::run_tests()
{
	printf("\nTesting Host Matrix.\n");

	// Verify we can get a zero matrix, should be hand checked
	printf("\nCreating a matrix of zeros of size 2x2:\n");
	Matrix<host_vector<float>> hmat0(2);
	thrust::fill(hmat0.vector.begin(), hmat0.vector.end(), 0.0);
	hmat0.print();
	
	//// Verify we can set values, should be hand
	printf("\nSet(0,0) = 1, (0, 1) = 2, (1, 0) = 3, (1, 1) = 4:\n");
	hmat0.set(0, 0, 1.0);
	hmat0.set(0, 1, 2.0);
	hmat0.set(1, 0, 3.0);
	hmat0.set(1, 1, 4.0);
	hmat0.print();

	printf("\nGet(0,0) (0, 1) (1, 0) (1, 1):\n");
	printf("%.1f ", hmat0.get(0,0));
	printf("%.1f", hmat0.get(0,1));
	printf("\n%.1f ", hmat0.get(1, 0));
	printf("%.1f\n", hmat0.get(1,1));

	// Setup alpha and beta yo
	float alpha = 2.0;
	float beta = 0.5;

	printf("\nTesting Host Matrix-Vector operations.\n");

	printf("\ngemv(A, x, y, alpha, beta, false):\n");
	host_vector<float> hvec0(2);
	hvec0[0] = -1.0;
	hvec0[1] = 0.0;

	host_vector<float> hvec1(2);
	hvec1[0] = 1.0;
	hvec1[1] = 1.5;

	printf("%.1f\n*", alpha);
	hmat0.print();
	printf("*\n%.1f %.1f\n+\n", (float)hvec0[0], (float)hvec0[1]);
	printf("%.1f\n*\n%.1f %.1f\n=\n", beta, (float)hvec1[0], (float)hvec1[1]);

	if(blas::gemv(hmat0, hvec0, hvec1, alpha, beta, false) != 0)
	{
		printf("Failure to execute host gemv.");
		return false;
	}

	printf("%.1f %.1f\n", (float)hvec1[0], (float)hvec1[1]);

	if(!(abs(hvec1[0] + 1.5) < Threshold &&
		abs(hvec1[1] + 5.25) < Threshold)
	  )
	{
		printf("Executed gemv but received an incorrect result.");
		return false;
	}

	printf("\ngemv(A, x, y, true):\n");
	printf("%.1f %.1f\n*", (float)hvec0[0], (float)hvec0[1]);
	hmat0.print();
	printf("=\n");

	if(blas::gemv(hmat0, hvec0, hvec1, true) != 0)
	{
		printf("Failed to execute host gemv.");
		return false;
	}

	printf("%.1f %.1f\n", (float)hvec1[0], (float)hvec1[1]);

	if(!(abs(hvec1[0] + 1.0) < Threshold &&
		abs(hvec1[1] + 2.0) < Threshold)
	  )
	{
		printf("Executed gemv but received an incorrect result.");
		return false;
	}

	printf("\nger(x, y, A, alpha):\n");
	printf("%.1f %.1f\nX\n%.1f %.1f\n+\n", (float)hvec0[0], (float)hvec0[1], (float)hvec1[0], (float)hvec1[1]);
	hmat0.print();
	printf("=");
	
	if(blas::ger(hvec0, hvec1, hmat0, alpha) != 0)
	{
		printf("Failed to execute ger.");
		return false;
	}

	hmat0.print();

	if(!(abs(hmat0.get(0, 0) - 3.0) < Threshold &&
		abs(hmat0.get(0, 1) - 6.0) < Threshold &&
		abs(hmat0.get(1, 0) - 3.0) < Threshold &&
		abs(hmat0.get(1, 1) - 4.0) < Threshold)
	  )
	{
		printf("Executed ger but received an incorrect result.");
		return false;
	}

	printf("\nTesting Host Vector-Vector operations.\n");

	printf("\nscal(x, alpha):\n");
	printf("%.1f\n*\n%.1f %.1f\n=\n", alpha, (float)hvec0[0], (float)hvec0[1]);

	if(blas::scal(hvec0, alpha) != 0)
	{
		printf("Failed to execute scal.");
		return false;
	}

	printf("%.1f %.1f\n", (float)hvec0[0], (float)hvec0[1]);

	if(!(abs(hvec0[0] + 2.0) < Threshold &&
		abs(hvec0[1] - 0.0) < Threshold)
	  )
	{
		printf("Executed scal but received an incorrect result.");
		return false;
	}

	printf("\ndot(x, y, result):\n");
	printf("%.1f %.1f\ndot\n%.1f %.1f\n=\n", (float)hvec0[0], (float)hvec0[1], (float)hvec1[0], (float)hvec1[1]);

	alpha = 0.0;
	if(blas::dot(hvec0, hvec1, alpha) != 0)
	{
		printf("Failed to execute host dot.");
		return false;
	}

	printf("%.1f\n", alpha);

	if(!(abs(alpha - 2.0) < Threshold))
	{
		printf("Executed dot but received an incorrect result.");
		return false;
	}

	printf("\naxpy(x, y, alpha):\n");
	printf("%.1f\n*\n%.1f %.1f\n+\n%.1f %.1f\n=\n", alpha, (float)hvec0[0], (float)hvec0[1], (float)hvec1[0], (float)hvec1[1]);

	if(blas::axpy(hvec0, hvec1, alpha) != 0)
	{
		printf("Failed to execute axpy.");
		return false;
	}

	printf("%.1f %.1f\n", (float)hvec1[0], (float)hvec1[1]);

	if(!(abs(hvec1[0] + 5.0) < Threshold &&
		abs(hvec1[1] + 2.0) < Threshold)
	  )
	{
		printf("Executed axpy but received an incorrect result.");
		return false;
	}

	printf("\nHost Passed!\n");

	printf("\nInitializing CUDA & Cublas\n");
	hipblasStatus_t stat = hipblasCreate(&blas::handle);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf("CUBLAS Init Failure.");
		return false;
	}

	printf("\nTesting Device Matrix.\n");

	// Verify we can get a zero matrix, should be hand checked
	printf("\nCreating a matrix of zeros of size 2x2:\n");
	Matrix<device_vector<float>> dmat0(2);
	thrust::fill(dmat0.vector.begin(), dmat0.vector.end(), 0.0);
	dmat0.print();
	
	//// Verify we can set values, should be hand
	printf("\nSet(0,0) = 1, (0, 1) = 2, (1, 0) = 3, (1, 1) = 4:\n");
	dmat0.set(0, 0, 1.0);
	dmat0.set(0, 1, 2.0);
	dmat0.set(1, 0, 3.0);
	dmat0.set(1, 1, 4.0);
	dmat0.print();

	printf("\nGet(0,0) (0, 1) (1, 0) (1, 1):\n");
	printf("%.1f ", dmat0.get(0,0));
	printf("%.1f", dmat0.get(0,1));
	printf("\n%.1f ", dmat0.get(1, 0));
	printf("%.1f\n", dmat0.get(1,1));

	// Reset alpha and beta yo
	alpha = 2.0;
	beta = 0.5;

	printf("\nTesting Device Matrix-Vector operations.\n");

	printf("\ngemv(A, x, y, alpha, beta, false):\n");
	device_vector<float> dvec0(2);
	dvec0[0] = -1.0;
	dvec0[1] = 0.0;

	device_vector<float> dvec1(2);
	dvec1[0] = 1.0;
	dvec1[1] = 1.5;

	printf("%.1f\n*\n", alpha);
	dmat0.print();
	printf("*\n%.1f %.1f\n+\n", (float)dvec0[0], (float)dvec0[1]);
	printf("%.1f\n*\n%.1f %.1f\n=\n", beta, (float)dvec1[0], (float)dvec1[1]);

	if(blas::gemv(dmat0, dvec0, dvec1, alpha, beta, false) != 0)
	{
		printf("Failure to execute device gemv.");
		return false;
	}

	printf("%.1f %.1f\n", (float)dvec1[0], (float)dvec1[1]);

	if(!(abs(dvec1[0] + 1.5) < Threshold &&
		abs(dvec1[1] + 5.25) < Threshold)
	  )
	{
		printf("Executed gemv but received an incorrect result.");
		return false;
	}

	printf("\ngemv(A, x, true):\n");
	printf("%.1f %.1f\n*", (float)dvec0[0], (float)dvec0[1]);
	dmat0.print();
	printf("=\n");

	if(blas::gemv(dmat0, dvec0, dvec1, true) != 0)
	{
		printf("Failed to execute gemv.");
		return false;
	}

	printf("%.1f %.1f\n", (float)dvec1[0], (float)dvec1[1]);

	if(!(abs(dvec1[0] + 1.0) < Threshold &&
		abs(dvec1[1] + 2.0) < Threshold)
	  )
	{
		printf("Executed gemv but received an incorrect result.");
		return false;
	}

	printf("\nger(x, y, A, alpha):\n");
	printf("%.1f %.1f\nX\n%.1f %.1f\n+", (float)dvec0[0], (float)dvec0[1], (float)dvec1[0], (float)dvec1[1]);
	dmat0.print();
	printf("=");
	
	if(blas::ger(dvec0, dvec1, dmat0, alpha) != 0)
	{
		printf("Failed to execute device ger.");
		return false;
	}

	dmat0.print();

	if(!(abs(dmat0.get(0, 0) - 3.0) < Threshold &&
		abs(dmat0.get(0, 1) - 6.0) < Threshold &&
		abs(dmat0.get(1, 0) - 3.0) < Threshold &&
		abs(dmat0.get(1, 1) - 4.0) < Threshold)
	  )
	{
		printf("Executed ger but received an incorrect result.");
		return false;
	}

	printf("\nTesting Device Vector-Vector operations.\n");

	printf("\nscal(x, alpha):\n");
	printf("%.1f\n*\n%.1f %.1f\n=\n", alpha, (float)dvec0[0], (float)dvec0[1]);

	if(blas::scal(dvec0, alpha) != 0)
	{
		printf("Failed to execute scal.");
		return false;
	}

	printf("%.1f %.1f\n", (float)dvec0[0], (float)dvec0[1]);

	if(!(abs(dvec0[0] + 2.0) < Threshold &&
		abs(dvec0[1] - 0.0) < Threshold)
	  )
	{
		printf("Executed scal but received an incorrect result.");
		return false;
	}

	printf("\ndot(x, y, result):\n");
	printf("%.1f %.1f\ndot\n%.1f %.1f\n=\n", (float)dvec0[0], (float)dvec0[1], (float)dvec1[0], (float)dvec1[1]);

	alpha = 0.0;
	if(blas::dot(dvec0, dvec1, alpha) != 0)
	{
		printf("Failed to execute dot.");
		return false;
	}

	printf("%.1f\n", alpha);

	if(!(abs(alpha - 2.0) < Threshold))
	{
		printf("Executed dot but received an incorrect result.");
		return false;
	}

	printf("\naxpy(x, y, alpha):\n");
	printf("%.1f\n*\n%.1f %.1f\n+\n%.1f %.1f\n=\n", alpha, (float)dvec0[0], (float)dvec0[1], (float)dvec1[0], (float)dvec1[1]);

	if(blas::axpy(dvec0, dvec1, alpha) != 0)
	{
		printf("Failed to execute axpy.");
		return false;
	}

	printf("%.1f %.1f\n", (float)dvec1[0], (float)dvec1[1]);

	if(!(abs(dvec1[0] + 5.0) < Threshold &&
		abs(dvec1[1] + 2.0) < Threshold)
	  )
	{
		printf("Executed axpy but received an incorrect result.");
		return false;
	}

	printf("\nDevice Passed!\n");

	printf("\nAll Tests Passed!\n");
	return true;
}